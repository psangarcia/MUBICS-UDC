
#include <hip/hip_runtime.h>
#include <stdio.h> 

__global__ void hello_kernel() {

    // calculate global thread identifier, note blockIdx.x=0 here
    const int thid = blockDim.x*blockIdx.x + threadIdx.x;

    // print a greeting message
    printf("Hola desde hilo %d!\n", thid);
}

int main (int argc, char * argv[]) {

    // set the ID of the CUDA device
    hipSetDevice(0);

    // invoke kernel using 16 threads executed in 1 thread block
    hello_kernel<<<1, 16>>>();

    // synchronize the GPU preventing premature termination
    hipDeviceSynchronize();
}
